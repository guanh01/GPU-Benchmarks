#include "hip/hip_runtime.h"

#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#define K 1
using namespace std;


#define trans_BLOCK_SIZE 16 

#define trans_GRID_X 160
#define trans_GRID_Y 480

#define trans_NBLOCKS (trans_GRID_X*trans_GRID_Y) 
texture<float,1,hipReadModeElementType> tex_1DA;
texture<float,2,hipReadModeElementType> tex_A;
surface<void,2> surf;
void computeGold( float* reference, float* idata,
            const unsigned int size_x, const unsigned int size_y )
{
  // transpose matrix
  for( unsigned int y = 0; y < size_y; ++y)
  {
    for( unsigned int x = 0; x < size_x; ++x)
    {
      reference[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

void trans_checkResults(float *h_idata, float *h_odata, int width, int height)
{
  for(unsigned int xIndex =0;xIndex<width;xIndex++){
   
   for (unsigned int yIndex =0;yIndex<height;yIndex++){
    if (xIndex < width && yIndex < height)
    {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
      if( h_odata[index_out] != h_idata[index_in]) {printf("failed!%d %d %f %f \n ",xIndex,yIndex,h_odata[index_out],h_idata[index_in]);return;}
    }
    }
    }
    printf("GOOD!trans passed\n");
    return;
}

__global__ void trans_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = trans_BLOCK_SIZE * blockIdx.x + threadIdx.x;//%trans_BLOCK_SIZE;
   unsigned int yIndex = trans_BLOCK_SIZE * blockIdx.y + threadIdx.y;///trans_BLOCK_SIZE;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
	   surf2Dwrite(idata[index_in],surf,yIndex*sizeof(float),xIndex);
	}
}

int main(int argc, char **argv) {
  hipSetDevice(2);
  srand(2013);

  const unsigned int trans_size_x = trans_GRID_X * trans_BLOCK_SIZE;
  const unsigned int trans_size_y = trans_GRID_Y * trans_BLOCK_SIZE;

  // size of memory required to store the matrix
  const unsigned int trans_mem_size = sizeof(float) * trans_size_x * trans_size_y;

  // allocate host memory
  float* h_trans_idata = (float*) malloc(trans_mem_size);
  // initalize the memory
  for( unsigned int i = 0; i < (trans_size_x * trans_size_y); ++i)
  {
      h_trans_idata[i] = (float) i;    // rand();
  }

  // allocate device memory
  float* d_trans_idata;
  float* d_trans_odata;
  hipMalloc( (void**) &d_trans_idata, trans_mem_size);
  hipMalloc( (void**) &d_trans_odata, trans_mem_size);

  // copy host memory to device
  hipMemcpy( d_trans_idata, h_trans_idata, trans_mem_size, hipMemcpyHostToDevice);

 hipChannelFormatDesc channelDescA =  hipCreateChannelDesc<float>();
 hipChannelFormatDesc forB= hipCreateChannelDesc<float>();//hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipArray* A_Array;
  hipArray* B_Array;
 hipMallocArray(&A_Array, &channelDescA, trans_size_x,trans_size_y);
 hipMallocArray(&B_Array, &forB,trans_size_y,trans_size_x,hipArraySurfaceLoadStore);
 hipMemcpyToArray(A_Array, 0, 0, h_trans_idata, trans_mem_size,
                      hipMemcpyHostToDevice);
 tex_A.addressMode[0] = hipAddressModeWrap;
    tex_A.addressMode[1] = hipAddressModeWrap;
    tex_A.filterMode     = hipFilterModePoint;
 hipBindTextureToArray(tex_A, A_Array, channelDescA);
 cudaBindSurfaceToArray(surf,B_Array,forB);
 hipBindTexture(0,tex_1DA,d_trans_idata,trans_mem_size);
  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;
 float* h_trans_odata = (float*) malloc(trans_mem_size);
  hipEventRecord(kernel_start, 0);
  // setup execution parameters
  dim3 trans_grid(trans_GRID_X, trans_GRID_Y, 1);
  dim3 trans_block(trans_BLOCK_SIZE, trans_BLOCK_SIZE, 1);
 

  trans_kernel<<<trans_grid, trans_block>>>(d_trans_odata, d_trans_idata, trans_size_x, trans_size_y);
//hipMemcpyFromArray(d_trans_odata,B_Array,0,0,trans_mem_size,hipMemcpyDeviceToDevice);
  //hipDeviceSynchronize();
//hipMemcpy( h_trans_odata, d_trans_odata, trans_mem_size, hipMemcpyDeviceToHost);
  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
//  hipMemcpy( h_trans_odata, d_trans_odata, trans_mem_size, hipMemcpyDeviceToHost);
  hipMemcpyFromArray(h_trans_odata,B_Array,0,0,trans_mem_size,hipMemcpyDeviceToHost);

  // check result
  trans_checkResults(h_trans_idata, h_trans_odata, trans_size_x, trans_size_y);


  return 0;
}

