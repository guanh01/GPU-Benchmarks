
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <list>
#include <map>
#include <math.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <algorithm>
#include <iterator>
#include <fstream>
#define K 1
using namespace std;


#define trans_BLOCK_SIZE 16

#define trans_GRID_X 160
#define trans_GRID_Y 480
#define trans_NBLOCKS (trans_GRID_X*trans_GRID_Y) 

void computeGold( float* reference, float* idata,
            const unsigned int size_x, const unsigned int size_y )
{
  // transpose matrix
  for( unsigned int y = 0; y < size_y; ++y)
  {
    for( unsigned int x = 0; x < size_x; ++x)
    {
      reference[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

void trans_checkResults(float *h_idata, float *h_odata, int width, int height)
{
  for(unsigned int xIndex =0;xIndex<width;xIndex++){
   
   for (unsigned int yIndex =0;yIndex<height;yIndex++){
    if (xIndex < width && yIndex < height)
    {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
      if( h_odata[index_out] != h_idata[index_in]) {printf("failed!%d %d %f %f \n ",xIndex,yIndex,h_odata[index_out],h_idata[index_in]);return;}
    }
    }
    }
    printf("GOOD!trans passed\n");
    return;
}

__global__ void trans_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
	   odata[index_out]=idata[index_in];
   }
}

int main(int argc, char **argv) {
  hipSetDevice(2);
  srand(2013);

  const unsigned int trans_size_x = trans_GRID_X * trans_BLOCK_SIZE;
  const unsigned int trans_size_y = trans_GRID_Y * trans_BLOCK_SIZE;

  // size of memory required to store the matrix
  const unsigned int trans_mem_size = sizeof(float) * trans_size_x * trans_size_y;

  // allocate host memory
  float* h_trans_idata = (float*) malloc(trans_mem_size);
  // initalize the memory
  for( unsigned int i = 0; i < (trans_size_x * trans_size_y); ++i)
  {
      h_trans_idata[i] = (float) i;    // rand();
  }

  // allocate device memory
  float* d_trans_idata;
  float* d_trans_odata;
  hipMalloc( (void**) &d_trans_idata, trans_mem_size);
  hipMalloc( (void**) &d_trans_odata, trans_mem_size);

  // copy host memory to device
  hipMemcpy( d_trans_idata, h_trans_idata, trans_mem_size, hipMemcpyHostToDevice);


  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float kernel_time = 0.0f;
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipEventRecord(kernel_start, 0);
  // setup execution parameters
  dim3 trans_grid(trans_GRID_X, trans_GRID_Y, 1);
  dim3 trans_block(trans_BLOCK_SIZE, trans_BLOCK_SIZE, 1);
 

  trans_kernel<<<trans_grid, trans_block>>>(d_trans_odata, d_trans_idata, trans_size_x, trans_size_y);

  hipDeviceSynchronize();

  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);

  // get elapsed time
  kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
  kernel_time *= 1.e-3; // Convert to seconds
  
  cout << "kernel exe time: " << kernel_time << endl;
  float* h_trans_odata = (float*) malloc(trans_mem_size);
  hipMemcpy( h_trans_odata, d_trans_odata, trans_mem_size, hipMemcpyDeviceToHost);


  // check result
  trans_checkResults(h_trans_idata, h_trans_odata, trans_size_x, trans_size_y);


  return 0;
}

